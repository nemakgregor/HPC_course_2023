#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define BLOCK_SIZE 16
#define FILTER_SIZE 100
#define FILTER_RADIUS FILTER_SIZE/2

__device__ unsigned char find_median(unsigned char* window) 
{
    for(int i = 0; i < FILTER_SIZE*FILTER_SIZE; i++) 
    {
        int min_idx = i;
        for(int j = i+1; j < FILTER_SIZE*FILTER_SIZE; j++) 
            if(window[j] < window[min_idx])
                min_idx = j;

        unsigned char temp = window[i];
        window[i] = window[min_idx];
        window[min_idx] = temp;
    }

    return window[FILTER_SIZE*FILTER_SIZE/2];
}

__global__ void median_filter_kernel(unsigned char* input, unsigned char* output, int width, int height) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx < width && idy < height) 
    {
        unsigned char filterValues[FILTER_SIZE * FILTER_SIZE];

        for (int i = 0; i < FILTER_SIZE; ++i) 
        {
            for (int j = 0; j < FILTER_SIZE; ++j) 
            {
                int currentX = min(max(idx + i - FILTER_RADIUS, 0), width - 1);
                int currentY = min(max(idy + j - FILTER_RADIUS, 0), height - 1);

                filterValues[i * FILTER_SIZE + j] = input[currentY * width + currentX];
            }
        }

        output[idy * width + idx] = find_median(filterValues);
    }
}

int main() 
{
    int width, height, channels;

    unsigned char* img = stbi_load("photo.jpg", &width, &height, &channels, 0);

    unsigned char* d_input;
    unsigned char* d_output;

    hipMalloc(&d_input, width * height * channels * sizeof(unsigned char));
    hipMalloc(&d_output, width * height * channels * sizeof(unsigned char));

    hipMemcpy(d_input, img, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    for(int c = 0; c < channels; c++)
        median_filter_kernel<<<grid, block>>>(d_input + c * width * height, d_output + c * width * height, width, height);

    hipMemcpy(img, d_output, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    stbi_write_jpg("photo_cartoon.jpg", width, height, channels, img, 100);

    hipFree(d_input);
    hipFree(d_output);
    stbi_image_free(img);

    return 0;
}
